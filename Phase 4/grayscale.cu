#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  if( i < cols && j < rows ) {
    g[ j * cols + i ] = (
      307 * rgb[ 3 * ( j * cols + i ) ]
      + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
      + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
    ) / 1024;
  }
}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC1, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;

  std::size_t size = m_in.cols * m_in.rows;
  std::size_t sizeRGB = 3 * m_in.cols * m_in.rows;


//  hipHostRegister(g.data(), size, hipHostRegisterDefault);

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, rows * cols );

  // Streams declaration.
  hipStream_t streams[ 2 ];

  // Creation.
  hipStreamCreate( &streams[ 0 ] );
  hipStreamCreate( &streams[ 1 ] );


  hipMemcpyAsync( rgb_d, rgb, sizeRGB/2, hipMemcpyHostToDevice, streams[ 0 ] );
  hipMemcpyAsync( rgb_d+sizeRGB/2, rgb+sizeRGB/2, sizeRGB/2, hipMemcpyHostToDevice, streams[ 1 ] );


  //hipMemcpyAsync( v1_d+size/2, v1+size/2, size/2 * sizeof(int), hipMemcpyHostToDevice, streams[ 1 ] );
  dim3 t( 32, 32 );
  dim3 be((( cols) / ((t.x - 2) + 1) ), (( rows ) / ((t.y - 2) + 1) ));
  // std::cout << "semi cols" << (m_in.cols / 2) << std::endl;
  // std::cout << "semi rows" << (m_in.rows / 2) << std::endl;
  // std::cout << "be x" << 3 * (( cols / 2 - 1) / (t.x + 1)) << std::endl;
  //
  // exit(0);

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );

  // One kernel is launched in each stream.
  grayscale<<< be, t, 0, streams[ 0 ] >>>( rgb_d, out, cols, rows);
  grayscale<<< be, t, 0, streams[ 1 ] >>>( rgb_d+sizeRGB/2, out+size/2, cols, rows);


  // Sending back the resulting vector by halves.
  hipMemcpyAsync( g.data(), out, size/2, hipMemcpyDeviceToHost, streams[ 0 ] );
  hipMemcpyAsync( g.data()+size/2, out+size/2, size/2, hipMemcpyDeviceToHost, streams[ 1 ] );

  // Synchronize everything.
  hipDeviceSynchronize();

  // Destroy streams.
  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);
  
  auto hipError_t = hipGetLastError();

  // Si pas d'erreur détectée dans le bordel ben on aura hipSuccess
  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }

  else {
    std::cout << "Aucune erreur" << std::endl;

  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";



  cv::imwrite( "outGrayscale.jpg", m_out );
  hipFree( rgb_d);
  //hipFree( g_d);
  hipFree ( out);


  return 0;
}
