#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void blur ( unsigned char * data,   unsigned char * out, std::size_t cols, std::size_t rows) {

  //auto i = blockIdx.x * (blockDim.x - 2) + threadIdx.x;
  //auto j = blockIdx.y * blockDim.y + threadIdx.y;

  auto i = blockIdx.x * (blockDim.x) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y) + threadIdx.y;

  if ( i > 0 && i < (cols - 1) && j > 0 && j < (rows - 1)) {

    for (auto c = 0; c < 3; ++c){

     auto gu =     data[((j - 1) * cols + i - 1) * 3 + c] +     data[((j - 1) * cols + i + 1) * 3 + c]
     +     data[( j      * cols + i - 1) * 3 + c] +     data[( j      * cols + i + 1) * 3 + c]
     +     data[((j + 1) * cols + i - 1) * 3 + c] +     data[((j + 1) * cols + i + 1) * 3 + c]
     +     data[(( j - 1) * cols + i) * 3 + c]     +     data[( j      * cols + i) * 3 + c]
     +     data[(( j + 1) * cols + i) * 3 + c];

     out[(j * cols + i) * 3 + c] = (gu / 9);
   }
  }

}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;

  std::size_t size = 3 * m_in.cols * m_in.rows;

//  hipHostRegister(g.data(), size, hipHostRegisterDefault);

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, 3 * rows * cols );

  // Streams declaration.
  hipStream_t streams[ 2 ];

  // Creation.
  hipStreamCreate( &streams[ 0 ] );
  hipStreamCreate( &streams[ 1 ] );

  hipMemcpyAsync( rgb_d, rgb, size/2, hipMemcpyHostToDevice, streams[ 0 ] );
  hipMemcpyAsync( rgb_d+size/2, rgb+size/2, size/2, hipMemcpyHostToDevice, streams[ 1 ] );

  dim3 t( 32, 32 );
  dim3 be( 3 * (( cols ) / ((t.x - 2) + 1) ), (( rows ) / ((t.y - 2) + 1) ));
  // dim3 t( 16, 16 );
  // dim3 be( 3 * 2 * (( cols ) / ((t.x - 2) + 1) ), (2 * ( rows ) / ((t.y - 2) + 1) ));
  // dim3 t( 4, 4 );
  // dim3 be( 3 * 8 * (( cols ) / ((t.x - 2) + 1) ), (8 * ( rows ) / ((t.y - 2) + 1) ));

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );

  // One kernel is launched in each stream.
  blur<<< be, t, 0, streams[ 0 ] >>>( rgb_d, out, cols, rows / 2 + 2);
  blur<<< be, t, 0, streams[ 1 ] >>>( rgb_d+size/2, out+size/2, cols, rows / 2 + 2);

  // Sending back the resulting vector by halves.
  hipMemcpyAsync( g.data(), out, size/2, hipMemcpyDeviceToHost, streams[ 0 ] );
  hipMemcpyAsync( g.data()+size/2, out+size/2, size/2, hipMemcpyDeviceToHost, streams[ 1 ] );

  // Synchronize everything.
  hipDeviceSynchronize();

  // Destroy streams.
  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);

  auto hipError_t = hipGetLastError();

  // Si pas d'erreur détectée dans le bordel ben on aura hipSuccess
  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }

  else {
    std::cout << "Aucune erreur" << std::endl;

  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";



  cv::imwrite( "outBlur.jpg", m_out );
  hipFree( rgb_d);
  //hipFree( g_d);
  hipFree ( out);


  return 0;
}
