#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void blurGauss ( unsigned char * data,   unsigned char * out, std::size_t cols, std::size_t rows) {

  //auto i = blockIdx.x * (blockDim.x - 2) + threadIdx.x;
  //auto j = blockIdx.y * blockDim.y + threadIdx.y;

  auto i = blockIdx.x * (blockDim.x) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y) + threadIdx.y;

  if ( i > 1 && i < (cols - 2) && j > 1 && j < (rows - 2)) {



      for (auto c = 0; c < 3; ++c){

        auto gu =  data[((j - 2) * cols + i - 2) * 3 + c] + 4 * data[((j - 2)  * cols + i - 1) * 3 + c]
        + 7 * data[((j - 2) * cols + i) * 3 + c]
        + 4 * data[((j - 2) * cols + i + 1) * 3 + c] + data[((j - 2)  * cols + i + 2) * 3 + c]
        + 4 * data[((j - 1) * cols + i  - 2) * 3 + c] + 7 * data[((j - 1)  * cols + i - 1) * 3 + c]
        + 26 * data[((j - 1) * cols + i) * 3 + c]
        + 7 * data[((j - 1) * cols + i + 1) * 3 + c] + 4 * data[((j - 1)  * cols + i + 2) * 3 + c]
        + 7 * data[((j) * cols + i - 2) * 3 + c] + 26 * data[((j)  * cols + i - 1) * 3 + c]
        + 41 * data[((j) * cols + i) * 3 + c]
        + 26 * data[((j) * cols + i + 1) * 3 + c] + 7 * data[((j)  * cols + i + 2) * 3 + c]
        + 4 * data[((j + 1) * cols + i - 2) * 3 + c] + 16 * data[((j + 1)  * cols + i - 1) * 3 + c]
        + 26 * data[((j + 1) * cols + i) * 3 + c]
        + 16 * data[((j + 1) * cols + i + 1) * 3 + c] + 4 * data[((j + 1)  * cols + i + 2) * 3 + c]
        + data[((j + 2) * cols + i - 2) * 3 + c] + 4 * data[((j + 2)  * cols + i - 1) * 3 + c]
        + 7 * data[((j + 2) * cols + i) * 3 + c]
        + 4 * data[((j + 2) * cols + i + 1) * 3 + c] + data[((j + 2)  * cols + i + 2) * 3 + c];

        out[(j * cols + i) * 3 + c] = (gu / 273);



      }
    }

}



__global__ void sobel ( unsigned char * data,   unsigned char * out, std::size_t cols, std::size_t rows) {

  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;


  if ( i > 0 && i < (cols - 1) && j > 0 && j < (rows - 1)) {



    for (auto c = 0; c < 3; ++c){

      auto h =   data[((j - 1) * cols + i - 1) * 3 + c] -     data[((j - 1) * cols + i + 1) * 3 + c]
      + 2 * data[( j      * cols + i - 1) * 3 + c] - 2 * data[( j      * cols + i + 1) * 3 + c]
      +     data[((j + 1) * cols + i - 1) * 3 + c] -     data[((j + 1) * cols + i + 1) * 3 + c];

      auto v =   data[((j - 1) * cols + i - 1) * 3 + c] -     data[((j + 1) * cols + i - 1) * 3 + c]
      + 2 * data[((j - 1) * cols + i    ) * 3 + c] - 2 * data[((j + 1) * cols + i    ) * 3 + c]
      +     data[((j - 1) * cols + i + 1) * 3 + c] -     data[((j + 1) * cols + i + 1) * 3 + c];

      auto res = h*h + v*v;
      res = res > 255*255 ? res = 255*255 : res;

      out[(j * cols + i) * 3 + c] = sqrt((float) res);



    }
  }

}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * g_d;
  unsigned char * out;

  std::size_t size = 3 * m_in.cols * m_in.rows;

//  hipHostRegister(g.data(), size, hipHostRegisterDefault);

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &g_d, 3 * rows * cols);
  hipMalloc( &out, 3 * rows * cols );

  // Streams declaration.
  hipStream_t streams[ 2 ];

  // Creation.
  hipStreamCreate( &streams[ 0 ] );
  hipStreamCreate( &streams[ 1 ] );

  hipMemcpyAsync( rgb_d, rgb, size/2, hipMemcpyHostToDevice, streams[ 0 ] );
  hipMemcpyAsync( rgb_d+size/2, rgb+size/2, size/2, hipMemcpyHostToDevice, streams[ 1 ] );

  //hipMemcpyAsync( v1_d+size/2, v1+size/2, size/2 * sizeof(int), hipMemcpyHostToDevice, streams[ 1 ] );
  // dim3 t( 32, 32 );
  // dim3 be( 3 * (( cols ) / ((t.x - 2) + 1) ), (( rows ) / ((t.y - 2) + 1) ));
  // dim3 t( 16, 16 );
  // dim3 be( 3 * 2 * (( cols ) / ((t.x - 2) + 1) ), (2 * ( rows ) / ((t.y - 2) + 1) ));
  dim3 t( 4, 4 );
  dim3 be( 3 * 8 * (( cols ) / ((t.x - 2) + 1) ), (8 * ( rows ) / ((t.y - 2) + 1) ));

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );

  // One kernel is launched in each stream.
  blurGauss<<< be, t, 0, streams[ 0 ] >>>( rgb_d, g_d, cols, rows / 2 + 4);
  blurGauss<<< be, t, 0, streams[ 1 ] >>>( rgb_d+size/2, g_d+size/2, cols, rows / 2);
  sobel<<< be, t, 0, streams[ 0 ] >>>( g_d, out, cols, rows / 2 + 4);
  sobel<<< be, t, 0, streams[ 1 ] >>>( g_d+size/2, out+size/2, cols, rows / 2);

  // Sending back the resulting vector by halves.
  hipMemcpyAsync( g.data(), out, size/2, hipMemcpyDeviceToHost, streams[ 0 ] );
  hipMemcpyAsync( g.data()+size/2, out+size/2, size/2, hipMemcpyDeviceToHost, streams[ 1 ] );

  // Synchronize everything.
  hipDeviceSynchronize();

  // Destroy streams.
  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);

  auto hipError_t = hipGetLastError();

  // Si pas d'erreur détectée dans le bordel ben on aura hipSuccess
  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }

  else {
    std::cout << "Aucune erreur" << std::endl;

  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";



  cv::imwrite( "outBlurGaussSobel.jpg", m_out );
  hipFree( rgb_d);
  //hipFree( g_d);
  hipFree ( out);


  return 0;
}
