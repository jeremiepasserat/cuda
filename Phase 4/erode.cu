#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void erode ( unsigned char * data,   unsigned char * out, std::size_t cols, std::size_t rows) {

  //auto i = blockIdx.x * (blockDim.x - 2) + threadIdx.x;
  //auto j = blockIdx.y * blockDim.y + threadIdx.y;

  auto i = blockIdx.x * (blockDim.x) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y) + threadIdx.y;

  if ( i > 0 && i < (cols - 1) && j > 0 && j < (rows - 1)) {

    for (auto c = 0; c < 3; ++c){

      auto hu = umax(data[((j - 1) * cols + i - 1) * 3 + c], data[((j - 1) * cols + i + 1) * 3 + c]);
      hu = umax(hu, data[(( j - 1) * cols + i) * 3 + c]);
      auto mu = umax(data[( j * cols + i - 1) * 3 + c],  data[( j * cols + i + 1) * 3 + c]);
      mu = umax(mu, data[( j * cols + i) * 3 + c]);
      auto bu = umax(data[((j + 1) * cols + i - 1) * 3 + c], data[((j + 1) * cols + i + 1) * 3 + c]);
      bu = umax(bu, data[((j + 1) * cols + i) * 3 + c]);

      auto res = umax (hu, mu);
      res = umax (res, bu);
      out[(j * cols + i) * 3 + c] = res;



    }
  }

}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;

  std::size_t size = 3 * m_in.cols * m_in.rows;

//  hipHostRegister(g.data(), size, hipHostRegisterDefault);

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, 3 * rows * cols );

  // Streams declaration.
  hipStream_t streams[ 2 ];

  // Creation.
  hipStreamCreate( &streams[ 0 ] );
  hipStreamCreate( &streams[ 1 ] );

  hipMemcpyAsync( rgb_d, rgb, size/2, hipMemcpyHostToDevice, streams[ 0 ] );
  hipMemcpyAsync( rgb_d+size/2, rgb+size/2, size/2, hipMemcpyHostToDevice, streams[ 1 ] );

  dim3 t( 32, 32 );
  dim3 be( 3 * (( cols ) / ((t.x - 2) + 1) ), (( rows ) / ((t.y - 2) + 1) ));
  dim3 t( 16, 16 );
  dim3 be( 3 * 2 * (( cols ) / ((t.x - 2) + 1) ), (2 * ( rows ) / ((t.y - 2) + 1) ));
  dim3 t( 4, 4 );
  dim3 be( 3 * 8 * (( cols ) / ((t.x - 2) + 1) ), (8 * ( rows ) / ((t.y - 2) + 1) ));

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );

  // One kernel is launched in each stream.
  erode<<< be, t, 0, streams[ 0 ] >>>( rgb_d, out, cols, rows / 2 + 2);
  erode<<< be, t, 0, streams[ 1 ] >>>( rgb_d+size/2, out+size/2, cols, rows / 2);

  // Sending back the resulting vector by halves.
  hipMemcpyAsync( g.data(), out, size/2, hipMemcpyDeviceToHost, streams[ 0 ] );
  hipMemcpyAsync( g.data()+size/2, out+size/2, size/2, hipMemcpyDeviceToHost, streams[ 1 ] );

  // Synchronize everything.
  hipDeviceSynchronize();

  // Destroy streams.
  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);

  auto hipError_t = hipGetLastError();

  // Si pas d'erreur détectée dans le bordel ben on aura hipSuccess
  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }

  else {
    std::cout << "Aucune erreur" << std::endl;

  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";



  cv::imwrite( "outErode.jpg", m_out );
  hipFree( rgb_d);
  //hipFree( g_d);
  hipFree ( out);


  return 0;
}
