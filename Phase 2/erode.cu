#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void erode ( unsigned char * data,   unsigned char * out, std::size_t cols, std::size_t rows) {

  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;


  if ( i > 1 && i < (cols - 2) && j > 1 && j < (rows - 2)) {



    for (auto c = 0; c < 3; ++c){

      auto hu = umax(data[((j - 1) * cols + i - 1) * 3 + c], data[((j - 1) * cols + i + 1) * 3 + c]);
      hu = umax(hu, data[(( j - 1) * cols + i) * 3 + c]);
      auto mu = umax(data[( j * cols + i - 1) * 3 + c],  data[( j * cols + i + 1) * 3 + c]);
      mu = umax(mu, data[( j * cols + i) * 3 + c]);
      auto bu = umax(data[((j + 1) * cols + i - 1) * 3 + c], data[((j + 1) * cols + i + 1) * 3 + c]);
      bu = umax(bu, data[((j + 1) * cols + i) * 3 + c]);

      auto res = umax (hu, mu);
      res = umax (res, bu);
      out[(j * cols + i) * 3 + c] = res;



    }
  }

}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, 3 * rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  dim3 t( 32, 32 );
  dim3 be(( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );

  // dim3 t( 16, 16 );
  // dim3 be( 3 * 2 * (( cols - 1) / t.x + 1 ), 2 * (( rows - 1 ) / t.y + 1 ));

  // dim3 t( 1, 1 );
  // dim3 be( 3 * 32 * (( cols - 1) / t.x + 1 ), 32 * (( rows - 1 ) / t.y + 1 ));

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start );

  erode<<< be, t >>>( rgb_d, out, cols, rows );

  hipMemcpy(g.data(), out, 3 * rows * cols, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  auto hipError_t = hipGetLastError();

  // Si pas d'erreur détectée dans le bordel ben on aura hipSuccess
  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }


  else {
    std::cout << "Aucune erreur" << std::endl;

  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";



  cv::imwrite( "outErode.jpg", m_out );
  hipFree( rgb_d);
  //hipFree( g_d);
  hipFree ( out);


  return 0;
}
