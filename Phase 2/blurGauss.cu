#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void blurGauss ( unsigned char * data,   unsigned char * out, std::size_t cols, std::size_t rows) {

  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;


  if ( i > 1 && i < (cols - 2) && j > 1 && j < (rows - 2)) {



    for (auto c = 0; c < 3; ++c){

      auto gu =  data[((j - 2) * cols + i - 2) * 3 + c] + 4 * data[((j - 2)  * cols + i - 1) * 3 + c]
      + 7 * data[((j - 2) * cols + i) * 3 + c]
      + 4 * data[((j - 2) * cols + i + 1) * 3 + c] + data[((j - 2)  * cols + i + 2) * 3 + c]
      + 4 * data[((j - 1) * cols + i  - 2) * 3 + c] + 7 * data[((j - 1)  * cols + i - 1) * 3 + c]
      + 26 * data[((j - 1) * cols + i) * 3 + c]
      + 7 * data[((j - 1) * cols + i + 1) * 3 + c] + 4 * data[((j - 1)  * cols + i + 2) * 3 + c]
      + 7 * data[((j) * cols + i - 2) * 3 + c] + 26 * data[((j)  * cols + i - 1) * 3 + c]
      + 41 * data[((j) * cols + i) * 3 + c]
      + 26 * data[((j) * cols + i + 1) * 3 + c] + 7 * data[((j)  * cols + i + 2) * 3 + c]
      + 4 * data[((j + 1) * cols + i - 2) * 3 + c] + 16 * data[((j + 1)  * cols + i - 1) * 3 + c]
      + 26 * data[((j + 1) * cols + i) * 3 + c]
      + 16 * data[((j + 1) * cols + i + 1) * 3 + c] + 4 * data[((j + 1)  * cols + i + 2) * 3 + c]
      + data[((j + 2) * cols + i - 2) * 3 + c] + 4 * data[((j + 2)  * cols + i - 1) * 3 + c]
      + 7 * data[((j + 2) * cols + i) * 3 + c]
      + 4 * data[((j + 2) * cols + i + 1) * 3 + c] + data[((j + 2)  * cols + i + 2) * 3 + c];

      out[(j * cols + i) * 3 + c] = (gu / 273);



    }
  }

}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, 3 * rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );

  // dim3 t( 32, 32 );
  // dim3 be(( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );

  dim3 t( 16, 16 );
  dim3 be( 3 * 2 * (( cols - 1) / t.x + 1 ), 2 * (( rows - 1 ) / t.y + 1 ));

  // dim3 t( 4, 4 );
  // dim3 be( 3 * 8 * (( cols - 1) / t.x + 1 ), 8 * (( rows - 1 ) / t.y + 1 ));

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start );

  blurGauss<<< be, t >>>( rgb_d, out, cols, rows );

  hipMemcpy(g.data(), out, 3 * rows * cols, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  auto hipError_t = hipGetLastError();

  // Si pas d'erreur détectée dans le bordel ben on aura hipSuccess
  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }


  else {
    std::cout << "Aucune erreur" << std::endl;

  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";



  cv::imwrite( "outBlurGauss.jpg", m_out );
  hipFree( rgb_d);
  //hipFree( g_d);
  hipFree ( out);


  return 0;
}
