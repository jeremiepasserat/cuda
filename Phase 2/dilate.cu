#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void dilate ( unsigned char * data,   unsigned char * out, std::size_t cols, std::size_t rows) {

  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;


  if ( i > 0 && i < (cols - 1) && j > 0 && j < (rows - 1)) {



    for (auto c = 0; c < 3; ++c){

      auto hu = umin(data[((j - 1) * cols + i - 1) * 3 + c], data[((j - 1) * cols + i + 1) * 3 + c]);
      hu = umin(hu, data[(( j - 1) * cols + i) * 3 + c]);
      auto mu = umin(data[( j * cols + i - 1) * 3 + c],  data[( j * cols + i + 1) * 3 + c]);
      mu = umin(mu, data[( j * cols + i) * 3 + c]);
      auto bu = umin(data[((j + 1) * cols + i - 1) * 3 + c], data[((j + 1) * cols + i + 1) * 3 + c]);
      bu = umin(bu, data[((j + 1) * cols + i) * 3 + c]);

      auto res = umin (hu, mu);
      res = umin (res, bu);
      out[(j * cols + i) * 3 + c] = res;



    }
  }

}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, 3 * rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  // dim3 t( 32, 32 );
  // dim3 be( 3 * (( cols - 1) / t.x + 1 ), (( rows - 1 ) / t.y + 1 ));

  dim3 t( 16, 16 );
  dim3 be( 3 * 2 * (( cols - 1) / t.x + 1 ), 2 * (( rows - 1 ) / t.y + 1 ));

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start );

  dilate<<< be, t >>>( rgb_d, out, cols, rows );

  hipMemcpy(g.data(), out, 3 * rows * cols, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  auto hipError_t = hipGetLastError();

  // Si pas d'erreur détectée dans le bordel ben on aura hipSuccess
  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }


  else {
    std::cout << "Aucune erreur" << std::endl;

  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";



  cv::imwrite( "outDilate.jpg", m_out );
  hipFree( rgb_d);
  //hipFree( g_d);
  hipFree ( out);


  return 0;
}
