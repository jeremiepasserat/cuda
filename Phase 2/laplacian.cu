#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void laplacian ( unsigned char * data,   unsigned char * out, std::size_t cols, std::size_t rows) {

  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;


  if ( i > 0 && i < (cols - 1) && j > 0 && j < (rows - 1)) {



    for (auto c = 0; c < 3; ++c){

      auto gu =     data[((j - 1) * cols + i - 1) * 3 + c] * 0 +     data[((j - 1) * cols + i + 1) * 3 + c] * 0
      +     data[( j      * cols + i - 1) * 3 + c] * -1 +     data[( j      * cols + i + 1) * 3 + c] * -1
      +     data[((j + 1) * cols + i - 1) * 3 + c] * 0 +     data[((j + 1) * cols + i + 1) * 3 + c] * 0
      +     data[(( j - 1) * cols + i) * 3 + c] * -1     +     data[( j      * cols + i) * 3 + c] * 4
      +     data[(( j + 1) * cols + i) * 3 + c] * -1;

      out[(j * cols + i) * 3 + c] = (gu / 9);



    }
  }

}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, 3 * rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  dim3 t( 32, 32 );
  dim3 be(( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );

  // dim3 t( 16, 16 );
  // dim3 be( 3 * 2 * (( cols - 1) / t.x + 1 ), 2 * (( rows - 1 ) / t.y + 1 ));

  // dim3 t( 4, 4 );
  // dim3 be( 3 * 8 * (( cols - 1) / t.x + 1 ), 8 * (( rows - 1 ) / t.y + 1 ));

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start );

  laplacian<<< be, t >>>( rgb_d, out, cols, rows );

  hipMemcpy(g.data(), out, 3 * rows * cols, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  auto hipError_t = hipGetLastError();

  // Si pas d'erreur détectée dans le bordel ben on aura hipSuccess
  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }

  else {
    std::cout << "Aucune erreur" << std::endl;

  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";



  cv::imwrite( "outLaplacian.jpg", m_out );
  hipFree( rgb_d);
  //hipFree( g_d);
  hipFree ( out);


  return 0;
}
