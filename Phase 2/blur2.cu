#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void blur2 ( unsigned char * data,   unsigned char * r,  unsigned char * g,  unsigned char * b, std::size_t cols, std::size_t rows) {

  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;

  if ( i > 0 && i < (cols - 1) && j > 0 && j < (rows - 1)) {

    for (auto c = 0; c < 3; ++c){

      auto gu =     data[((j - 1) * cols + i - 1) * 3 + c] +     data[((j - 1) * cols + i + 1) * 3 + c]
      +     data[( j      * cols + i - 1) * 3 + c] +     data[( j      * cols + i + 1) * 3 + c]
      +     data[((j + 1) * cols + i - 1) * 3 + c] +     data[((j + 1) * cols + i + 1) * 3 + c]
      +     data[(( j - 1) * cols + i) * 3 + c]     +     data[( j      * cols + i) * 3 + c]
      +     data[(( j + 1) * cols + i) * 3 + c];


      if (c == 2)
      b[(j * cols + i)] = (gu/9);

      else if (c == 1)

      g[(j * cols + i)] =(gu/9);

      else if (c == 0)

      r[(j * cols + i)] = (gu/9);
    }
  }
}


int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > r_out( rows * cols );
  cv::Mat r_mat( rows, cols, CV_8UC1, r_out.data() );
  std::vector< unsigned char > g_out( rows * cols );
  cv::Mat g_mat( rows, cols, CV_8UC1, g_out.data() );
  std::vector< unsigned char > b_out( rows * cols );
  cv::Mat b_mat( rows, cols, CV_8UC1, b_out.data() );
  std::vector<cv::Mat> toMerge;
  cv::Mat m_out;
  unsigned char * rgb_d;
  unsigned char * r;
  unsigned char * g;
  unsigned char * b;

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &r, rows * cols );
  hipMalloc( &g, rows * cols );
  hipMalloc( &b, rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  //dim3 t( 32, 32 );
  //dim3 be(( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );

  dim3 t( 16, 16 );
  dim3 be( 3 * 2 * (( cols - 1) / t.x + 1 ), 2 * (( rows - 1 ) / t.y + 1 ));

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start );

  blur2<<< be, t >>>( rgb_d, r, g, b, cols, rows );

  hipMemcpy(r_out.data(), r, rows * cols, hipMemcpyDeviceToHost);
  hipMemcpy(g_out.data(), g, rows * cols, hipMemcpyDeviceToHost);
  hipMemcpy(b_out.data(), b, rows * cols, hipMemcpyDeviceToHost);

  toMerge.push_back(r_mat);
  toMerge.push_back(g_mat);
  toMerge.push_back(b_mat);

  cv::merge(toMerge, m_out);

  hipDeviceSynchronize();

  auto hipError_t = hipGetLastError();

  // Si pas d'erreur détectée dans le bordel ben on aura hipSuccess
  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }

  else {
    std::cout << "Aucune erreur" << std::endl;

  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";



  cv::imwrite( "outBlur2.jpg", m_out);
  hipFree( rgb_d);
  //hipFree( g_d);
  hipFree ( r);
  hipFree ( g);
  hipFree ( b);


  return 0;
}
