#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void blurGaussShared ( unsigned char * data,   unsigned char * out, std::size_t w, std::size_t h) {

  auto i = blockIdx.x * (blockDim.x-4) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-4) + threadIdx.y;

  auto li = threadIdx.x;
  auto lj = threadIdx.y;


  extern __shared__ unsigned char sh[];

  if( i < w && j < h ) {

    // on s'occupe du rouge
    sh[3 * (lj * blockDim.x + li) ] = data[ 3 * ( j * w + i ) ];
    sh[3 * (lj * blockDim.x + li) + 1 ] = data[ 3 * ( j * w + i ) + 1];
    sh[3 * (lj * blockDim.x + li) + 2 ] = data[ 3 * ( j * w + i ) + 2 ];

    __syncthreads();

    auto ww = blockDim.x;

    if( li > 1 && li < (blockDim.x - 2) && lj > 1 && lj < (blockDim.y - 2) )
    {
      for (auto c = 0; c < 3; ++c){

          /*
            1 : 1 - 4 - 7 - 4 - 1
            2 : 4 - 16 - 26 - 16 - 4
            3 : 7 - 26 - 41 - 26 - 7
            4 : 4 - 16 - 26 - 16 - 4
            5 : 1 - 4 - 7 - 4 - 1

          */

          auto gu = sh[(( lj - 2) * ww + li - 2) * 3 + c] + sh[(( lj - 2) * ww + li + 2) * 3 + c]
          +     4 * sh[(( lj - 2) * ww + li - 1) * 3 + c] + 4 * sh[(( lj - 2) * ww + li + 1) * 3 + c]
          +     7 *sh[(( lj - 2) * ww + li) * 3 + c]
          +     4 *sh[(( lj - 1) * ww + li - 2) * 3 + c] + 4 * sh[(( lj - 1) * ww + li + 2) * 3 + c]
          +     16 * sh[(( lj - 1) * ww + li - 1) * 3 + c] + 16 * sh[(( lj - 1) * ww + li + 1) * 3 + c]
          +     26 * sh[(( lj - 1) * ww + li) * 3 + c]
          +     7 * sh[(( lj ) * ww + li - 1) * 3 + c] + 7 * sh[(( lj ) * ww + li + 1) * 3 + c]
          +     26 * sh[(( lj ) * ww + li - 2) * 3 + c] + 26 * sh[(( lj ) * ww + li + 2) * 3 + c]
          +     41 * sh[(( lj ) * ww + li) * 3 + c]
          +     4 * sh[(( lj + 1) * ww + li - 1) * 3 + c] + 4 * sh[(( lj + 1) * ww + li + 1) * 3 + c]
          +     16 * sh[(( lj + 1) * ww + li - 2) * 3 + c] + 16 * sh[(( lj + 1) * ww + li + 2) * 3 + c]
          +     26 * sh[(( lj + 1) * ww + li) * 3 + c]
          +     sh[(( lj + 2) * ww + li - 1) * 3 + c] + sh[(( lj + 2) * ww + li + 1) * 3 + c]
          +     4 * sh[(( lj + 2) * ww + li - 2) * 3 + c] + 4 * sh[(( lj + 2) * ww + li + 2) * 3 + c]
          +     7 * sh[(( lj + 2) * ww + li) * 3 + c];

          out[(j * w + i) * 3 + c] = (gu / 273);

      }

    }
  }
}


int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, 3 * rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  dim3 t( 32, 32 );
  dim3 be( 3 * (( cols - 1) / t.x + 1 ), (( rows - 1 ) / t.y + 1 ));
  dim3 bu( 3 * (( cols - 1) / (t.x-4) + 1) , ( rows - 1 ) / (t.y-4) + 1 );

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start );

  blurGaussShared<<< bu, t, 3*t.x*t.y >>>( rgb_d, out, cols, rows );
  hipMemcpy(g.data(), out, 3 * rows * cols, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  auto hipError_t = hipGetLastError();

  // Si pas d'erreur détectée dans le bordel ben on aura hipSuccess
  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }

  else {
    std::cout << "Aucune erreur" << std::endl;

  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";

  cv::imwrite( "outBlurGaussShared.jpg", m_out );
  hipFree( rgb_d);
  hipFree ( out);


  return 0;
}
