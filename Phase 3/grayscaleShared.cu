#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void grayscaleShared ( unsigned char * in,   unsigned char * out, std::size_t w, std::size_t h) {

  auto i = blockIdx.x * (blockDim.x-2) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-2) + threadIdx.y;

  auto li = threadIdx.x;
  auto lj = threadIdx.y;

  extern __shared__ unsigned char sh[];

  if( i < w && j < h ) {

    sh[ (lj * blockDim.x + li) ] = (
      307 * in[ 3 * ( j * w + i ) ]
      + 604 * in[ 3 * ( j * w + i ) + 1 ]
      + 113 * in[  3 * ( j * w + i ) + 2 ]
    ) / 1024;

    __syncthreads();

    out[(j * w + i)] = sh[(lj * blockDim.x + li)];

  }
}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC1, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  // dim3 t( 32, 32 );
  // dim3 bu(  (( cols - 1) / (t.x-2) + 1) , ( rows - 1 ) / (t.y-2) + 1 );

  // dim3 t( 16, 16 );
  // dim3 bu(  2 * (( cols - 1) / (t.x-2) + 1) , (2 * rows - 1 ) / (t.y-2) + 1 );

  dim3 t( 4, 4 );
  dim3 bu(  8 *(( cols - 1) / (t.x-2) + 1) , (8 * rows - 1 ) / (t.y-2) + 1 );

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start );

  grayscaleShared<<< bu, t, t.x*t.y >>>( rgb_d, out, cols, rows );
  hipMemcpy(g.data(), out, rows * cols, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  auto hipError_t = hipGetLastError();

  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }

  else {
    std::cout << "Aucune erreur" << std::endl;

  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";

  cv::imwrite( "outGrayscaleShared.jpg", m_out );
  hipFree( rgb_d);
  hipFree ( out);


  return 0;
}
