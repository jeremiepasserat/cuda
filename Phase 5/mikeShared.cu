#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void mikeShared ( unsigned char * data,   unsigned char * out, std::size_t w, std::size_t h) {

  auto i = blockIdx.x * (blockDim.x-2) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-2) + threadIdx.y;

  auto li = threadIdx.x;
  auto lj = threadIdx.y;

  extern __shared__ unsigned char sh[];

  if( i < w && j < h ) {

    // on s'occupe du rouge
    sh[3 * (lj * blockDim.x + li) ] = data[ 3 * ( j * w + i ) ];
    sh[3 * (lj * blockDim.x + li) + 1 ] = data[ 3 * ( j * w + i ) + 1];
    sh[3 * (lj * blockDim.x + li) + 2 ] = data[ 3 * ( j * w + i ) + 2 ];

    __syncthreads();

    auto ww = blockDim.x;

    if( li > 0 && li < (blockDim.x - 1) && lj > 0 && lj < (blockDim.y - 1) )
    {
      for (auto c = 0; c < 3; ++c){

        auto gu =     sh[((lj - 1) * ww + li - 1) * 3 + c] * 2 + sh[((lj - 1) * ww + li + 1) * 3 + c]
    +     sh[( lj      * ww + li - 1) * 3 + c] * 22 +     sh[( lj      * ww + li + 1) * 3 + c] * -22
    +     sh[((lj + 1) * ww + li - 1) * 3 + c]  +     sh[((lj + 1) * ww + li + 1) * 3 + c] * -2
    +     sh[(( lj - 1) * ww + li) * 3 + c] * 22   +  2 *   sh[( lj      * ww + li) * 3 + c]
    +     sh[(( lj + 1) * ww + li) * 3 + c] * -22;

          out[(j * w + i) * 3 + c] = (gu / 9);

      }

    }
  }
}
int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;

  std::size_t size = 3 * m_in.cols * m_in.rows;

//  hipHostRegister(g.data(), size, hipHostRegisterDefault);

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, 3 * rows * cols );

  // Streams declaration.
  hipStream_t streams[ 2 ];

  // Creation.
  hipStreamCreate( &streams[ 0 ] );
  hipStreamCreate( &streams[ 1 ] );

  hipMemcpyAsync( rgb_d, rgb, size/2, hipMemcpyHostToDevice, streams[ 0 ] );
  hipMemcpyAsync( rgb_d+size/2, rgb+size/2, size/2, hipMemcpyHostToDevice, streams[ 1 ] );

  // dim3 t( 32, 32 );
  // dim3 be( 3 * (( cols ) / ((t.x - 2) + 1) ), (( rows ) / ((t.y - 2) + 1) ));
  // dim3 t( 16, 16 );
  // dim3 be( 3 * 2 * (( cols ) / ((t.x - 2) + 1) ), ( 2 *  rows  / ((t.y - 2) + 1) ));

  dim3 t( 4, 4 );
  dim3 be( 3 * 8 * (( cols ) / ((t.x - 2) + 1) ), ( 8 *  rows  / ((t.y - 2) + 1) ));

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );

  // One kernel is launched in each stream.
  mikeShared<<< be, t, 3 * t.x * t.y, streams[ 0 ] >>>( rgb_d, out, cols, rows / 2 + 2);
  mikeShared<<< be, t, 3 * t.x * t.y, streams[ 1 ] >>>( rgb_d+size/2, out+size/2, cols, rows / 2);

  // Sending back the resulting vector by halves.
  hipMemcpyAsync( g.data(), out, size/2, hipMemcpyDeviceToHost, streams[ 0 ] );
  hipMemcpyAsync( g.data()+size/2, out+size/2, size/2, hipMemcpyDeviceToHost, streams[ 1 ] );

  // Synchronize everything.
  hipDeviceSynchronize();

  // Destroy streams.
  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);

  auto hipError_t = hipGetLastError();

  // Si pas d'erreur détectée dans le bordel ben on aura hipSuccess
  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }

  else {
    std::cout << "Aucune erreur" << std::endl;

  }



  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";



  cv::imwrite( "outMikeShared.jpg", m_out );
  hipFree( rgb_d);
  //hipFree( g_d);
  hipFree ( out);


  return 0;
}
