#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void blurGaussShared ( unsigned char * data,   unsigned char * out, std::size_t w, std::size_t h) {

  auto i = blockIdx.x * (blockDim.x-4) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-4) + threadIdx.y;

  auto li = threadIdx.x;
  auto lj = threadIdx.y;


  extern __shared__ unsigned char sh[];

  if( i < w && j < h ) {

    // on s'occupe du rouge
    sh[3 * (lj * blockDim.x + li) ] = data[ 3 * ( j * w + i ) ];
    sh[3 * (lj * blockDim.x + li) + 1 ] = data[ 3 * ( j * w + i ) + 1];
    sh[3 * (lj * blockDim.x + li) + 2 ] = data[ 3 * ( j * w + i ) + 2 ];

    __syncthreads();

    auto ww = blockDim.x;

    if( li > 1 && li < (blockDim.x - 2) && lj > 1 && lj < (blockDim.y - 2) )
    {
      for (auto c = 0; c < 3; ++c){

          /*
            1 : 1 - 4 - 7 - 4 - 1
            2 : 4 - 16 - 26 - 16 - 4
            3 : 7 - 26 - 41 - 26 - 7
            4 : 4 - 16 - 26 - 16 - 4
            5 : 1 - 4 - 7 - 4 - 1

          */

          auto gu = sh[(( lj - 2) * ww + li - 2) * 3 + c] + sh[(( lj - 2) * ww + li + 2) * 3 + c]
          +     4 * sh[(( lj - 2) * ww + li - 1) * 3 + c] + 4 * sh[(( lj - 2) * ww + li + 1) * 3 + c]
          +     7 *sh[(( lj - 2) * ww + li) * 3 + c]
          +     4 *sh[(( lj - 1) * ww + li - 2) * 3 + c] + 4 * sh[(( lj - 1) * ww + li + 2) * 3 + c]
          +     16 * sh[(( lj - 1) * ww + li - 1) * 3 + c] + 16 * sh[(( lj - 1) * ww + li + 1) * 3 + c]
          +     26 * sh[(( lj - 1) * ww + li) * 3 + c]
          +     7 * sh[(( lj ) * ww + li - 1) * 3 + c] + 7 * sh[(( lj ) * ww + li + 1) * 3 + c]
          +     26 * sh[(( lj ) * ww + li - 2) * 3 + c] + 26 * sh[(( lj ) * ww + li + 2) * 3 + c]
          +     41 * sh[(( lj ) * ww + li) * 3 + c]
          +     4 * sh[(( lj + 1) * ww + li - 1) * 3 + c] + 4 * sh[(( lj + 1) * ww + li + 1) * 3 + c]
          +     16 * sh[(( lj + 1) * ww + li - 2) * 3 + c] + 16 * sh[(( lj + 1) * ww + li + 2) * 3 + c]
          +     26 * sh[(( lj + 1) * ww + li) * 3 + c]
          +     sh[(( lj + 2) * ww + li - 1) * 3 + c] + sh[(( lj + 2) * ww + li + 1) * 3 + c]
          +     4 * sh[(( lj + 2) * ww + li - 2) * 3 + c] + 4 * sh[(( lj + 2) * ww + li + 2) * 3 + c]
          +     7 * sh[(( lj + 2) * ww + li) * 3 + c];

          out[(j * w + i) * 3 + c] = (gu / 273);

      }

    }
  }
}



int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;

  std::size_t size = 3 * m_in.cols * m_in.rows;

//  hipHostRegister(g.data(), size, hipHostRegisterDefault);

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, 3 * rows * cols );

  // Streams declaration.
  hipStream_t streams[ 2 ];

  // Creation.
  hipStreamCreate( &streams[ 0 ] );
  hipStreamCreate( &streams[ 1 ] );

  hipMemcpyAsync( rgb_d, rgb, size/2, hipMemcpyHostToDevice, streams[ 0 ] );
  hipMemcpyAsync( rgb_d+size/2, rgb+size/2, size/2, hipMemcpyHostToDevice, streams[ 1 ] );


  dim3 t( 32, 32 );
  dim3 be( 3 * (( cols ) / ((t.x - 4) + 1) ), (( rows ) / ((t.y - 4) + 1) ));

  // dim3 t( 16, 16 );
  // dim3 be( 3 * 2 * (( cols ) / ((t.x - 4) + 1) ), ( 2 *  rows  / ((t.y - 4) + 1) ));

  // dim3 t( 4, 4 );
  // dim3 be( 3 * 8 * (( cols ) / ((t.x - 4) + 1) ), ( 8 *  rows  / ((t.y - 4) + 1) ));


  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );

  // One kernel is launched in each stream.
  blurGaussShared<<< be, t, 3 * t.x * t.y, streams[ 0 ] >>>( rgb_d, out, cols, rows / 2 + 8);
  blurGaussShared<<< be, t, 3 * t.x * t.y, streams[ 1 ] >>>( rgb_d+size/2, out+size/2, cols, rows / 2 + 8);

  // Sending back the resulting vector by halves.
  hipMemcpyAsync( g.data(), out, size/2, hipMemcpyDeviceToHost, streams[ 0 ] );
  hipMemcpyAsync( g.data()+size/2, out+size/2, size/2, hipMemcpyDeviceToHost, streams[ 1 ] );

  // Synchronize everything.
  hipDeviceSynchronize();

  // Destroy streams.
  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);

  auto hipError_t = hipGetLastError();

  // Si pas d'erreur détectée dans le bordel ben on aura hipSuccess
  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }

  else {
    std::cout << "Aucune erreur" << std::endl;

  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";



  cv::imwrite( "outBlurGaussShared.jpg", m_out );
  hipFree( rgb_d);
  //hipFree( g_d);
  hipFree ( out);


  return 0;
}
