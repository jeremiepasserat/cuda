#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void sobelShared ( unsigned char * data,   unsigned char * out, std::size_t w, std::size_t h) {

  auto i = blockIdx.x * (blockDim.x-2) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-2) + threadIdx.y;

  auto li = threadIdx.x;
  auto lj = threadIdx.y;

  extern __shared__ unsigned char sh[];

  if( i < w && j < h ) {

    // on s'occupe du rouge
    sh[3 * (lj * blockDim.x + li) ] = data[ 3 * ( j * w + i ) ];
    sh[3 * (lj * blockDim.x + li) + 1 ] = data[ 3 * ( j * w + i ) + 1];
    sh[3 * (lj * blockDim.x + li) + 2 ] = data[ 3 * ( j * w + i ) + 2 ];

    __syncthreads();

    auto ww = blockDim.x;

    if( li > 0 && li < (blockDim.x - 1) && lj > 0 && lj < (blockDim.y - 1) )
    {
      for (auto c = 0; c < 3; ++c){

        auto hh = sh[ ((lj-1)*ww + li - 1)* 3 + c ] - sh[ ((lj-1)*ww + li + 1) * 3 + c ]
        + 2 * sh[ (lj*ww + li - 1) * 3 + c ] - 2* sh[ (lj*ww+li+1) * 3 + c]
        + sh[ ((lj+1)*ww + li -1) * 3 + c] - sh[ ((lj+1)*ww +li + 1) * 3 + c];
        auto vv = sh[ ((lj-1)*ww + li - 1) * 3 + c ] - sh[ ((lj+1)*ww + li - 1) * 3 + c ]
        + 2 * sh[ ((lj-1)*ww + li) * 3 + c ] - 2* sh[ ((lj+1)*ww+li) * 3 + c ]
        + sh[ ((lj-1)*ww + li +1) * 3 + c] - sh[ ((lj+1)*ww +li + 1) * 3 + c];

        auto res = hh * hh + vv * vv;
        res = res > 255*255 ? res = 255*255 : res;
        out[ (j * w + i) * 3 + c ] = sqrt( (float)res );

      }

    }
  }
}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( 3 * rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC3, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;

  std::size_t size = 3 * m_in.cols * m_in.rows;

//  hipHostRegister(g.data(), size, hipHostRegisterDefault);

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, 3 * rows * cols );

  // Streams declaration.
  hipStream_t streams[ 4 ];

  // Creation.
  hipStreamCreate( &streams[ 0 ] );
  hipStreamCreate( &streams[ 1 ] );
  hipStreamCreate( &streams[ 2 ] );
  hipStreamCreate( &streams[ 3 ] );


  hipMemcpyAsync( rgb_d, rgb, size/4, hipMemcpyHostToDevice, streams[ 0 ] );
  hipMemcpyAsync( rgb_d+size/4, rgb+size/4, size/4, hipMemcpyHostToDevice, streams[ 1 ] );
  hipMemcpyAsync( rgb_d+size/2, rgb+size/2, size/4, hipMemcpyHostToDevice, streams[ 1 ] );
  hipMemcpyAsync( rgb_d+3*size/4, rgb+3*size/4, size/4, hipMemcpyHostToDevice, streams[ 1 ] );

  //hipMemcpyAsync( v1_d+size/2, v1+size/2, size/2 * sizeof(int), hipMemcpyHostToDevice, streams[ 1 ] );
  dim3 t( 32, 32 );
  dim3 be( 3 * (( cols / 2) / (t.x + 1) ), (( rows / 2) / (t.y + 1) ));
  dim3 bu( 3 * (( cols - 1) / (t.x-2) + 1) , ( rows - 1 ) / (t.y-2) + 1 );
  // std::cout << "semi cols" << (m_in.cols / 2) << std::endl;
  // std::cout << "semi rows" << (m_in.rows / 2) << std::endl;
  // std::cout << "be x" << 3 * (( cols / 2 - 1) / (t.x + 1)) << std::endl;
  //
  // exit(0);

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );

  // One kernel is launched in each stream.
  sobelShared<<< bu, t, 3 * t.x * t.y, streams[ 0 ] >>>( rgb_d, out, cols, rows/4 + 2);
  sobelShared<<< bu, t, 3 * t.x * t.y, streams[ 1 ] >>>( rgb_d+size/4, out+size/4, cols, rows/4 + 4);
  sobelShared<<< bu, t, 3 * t.x * t.y, streams[ 2 ] >>>( rgb_d+size/2, out+size/2, cols, rows/4 + 2);
  sobelShared<<< bu, t, 3 * t.x * t.y, streams[ 3 ] >>>( rgb_d+3*size/4, out+3*size/4, cols, rows/4 );

  // Sending back the resulting vector by halves.
  hipMemcpyAsync( g.data(), out, size/4, hipMemcpyDeviceToHost, streams[ 0 ] );
  hipMemcpyAsync( g.data()+size/4, out+size/4, size/4, hipMemcpyDeviceToHost, streams[ 1 ] );
  hipMemcpyAsync( g.data()+size/2, out+size/2, size/4, hipMemcpyDeviceToHost, streams[ 2 ] );
  hipMemcpyAsync( g.data()+3*size/4, out+3*size/4, size/4, hipMemcpyDeviceToHost, streams[ 3 ] );

  // Synchronize everything.
  hipDeviceSynchronize();

  // Destroy streams.
  hipDeviceSynchronize();

  auto hipError_t = hipGetLastError();

  // Si pas d'erreur détectée dans le bordel ben on aura hipSuccess
  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }

  else {
    std::cout << "Aucune erreur" << std::endl;

  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";



  cv::imwrite( "outSobelShared.jpg", m_out );
  hipFree( rgb_d);
  //hipFree( g_d);
  hipFree ( out);


  return 0;
}
