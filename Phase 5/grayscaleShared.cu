#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void grayscaleShared ( unsigned char * in,   unsigned char * out, std::size_t w, std::size_t h) {

  auto i = blockIdx.x * (blockDim.x-2) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-2) + threadIdx.y;

  auto li = threadIdx.x;
  auto lj = threadIdx.y;

  extern __shared__ unsigned char sh[];

  if( i < w && j < h ) {

    sh[ (lj * blockDim.x + li) ] = (
      307 * in[ 3 * ( j * w + i ) ]
      + 604 * in[ 3 * ( j * w + i ) + 1 ]
      + 113 * in[  3 * ( j * w + i ) + 2 ]
    ) / 1024;

    __syncthreads();

    out[(j * w + i)] = sh[(lj * blockDim.x + li)];

  }
}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC1, g.data() );
  unsigned char * rgb_d;
  unsigned char * out;

  std::size_t size = m_in.cols * m_in.rows;
  std::size_t sizeRGB = 3 * m_in.cols * m_in.rows;


//  hipHostRegister(g.data(), size, hipHostRegisterDefault);

  hipMalloc( &rgb_d, 3 * rows * cols);
  hipMalloc( &out, rows * cols );

  // Streams declaration.
  hipStream_t streams[ 2 ];

  // Creation.
  hipStreamCreate( &streams[ 0 ] );
  hipStreamCreate( &streams[ 1 ] );


  hipMemcpyAsync( rgb_d, rgb, sizeRGB/2, hipMemcpyHostToDevice, streams[ 0 ] );
  hipMemcpyAsync( rgb_d+sizeRGB/2, rgb+sizeRGB/2, sizeRGB/2, hipMemcpyHostToDevice, streams[ 1 ] );


  dim3 t( 32, 32 );
  dim3 be(  (( cols ) / ((t.x - 2) + 1) ), (( rows ) / ((t.y - 2) + 1) ));
  // dim3 t( 16, 16 );
  // dim3 be(  2 * (( cols ) / ((t.x - 2) + 1) ), ( 2 *  rows  / ((t.y - 2) + 1) ));

  // dim3 t( 4, 4 );
  // dim3 be(  8 * (( cols ) / ((t.x - 2) + 1) ), ( 8 *  rows  / ((t.y - 2) + 1) ));

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );

  // One kernel is launched in each stream.
  grayscaleShared<<< be, t, t.x * t.y, streams[ 0 ] >>>( rgb_d, out, cols + 2, rows);
  grayscaleShared<<< be, t, t.x * t.y, streams[ 1 ] >>>( rgb_d+sizeRGB/2, out+size/2, cols + 2, rows);


  // Sending back the resulting vector by halves.
  hipMemcpyAsync( g.data(), out, size/2, hipMemcpyDeviceToHost, streams[ 0 ] );
  hipMemcpyAsync( g.data()+size/2, out+size/2, size/2, hipMemcpyDeviceToHost, streams[ 1 ] );

  // Synchronize everything.
  hipDeviceSynchronize();

  // Destroy streams.
  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);

  auto hipError_t = hipGetLastError();

  // Si pas d'erreur détectée dans le bordel ben on aura hipSuccess
  if (hipError_t != hipSuccess){

    std::cout << hipGetErrorName(hipError_t) << std::endl;
    std::cout << hipGetErrorString(hipError_t) << std::endl;
  }

  else {
    std::cout << "Aucune erreur" << std::endl;

  }
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total: " << duration << "ms\n";



  cv::imwrite( "outGrayscaleShared.jpg", m_out );
  hipFree( rgb_d);
  //hipFree( g_d);
  hipFree ( out);


  return 0;
}
